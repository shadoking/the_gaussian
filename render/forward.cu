#include "hip/hip_runtime.h"
#include "forward.h"

__device__ void strConcat(char* dest, const char* src) {
    while (*dest) {
        dest++;
    }
    while ((*dest++ = *src++)) { }
}

__device__ void printMatrix3(Eigen::Matrix3f matrix, char* name) {
    strConcat(name, ":\n%f, %f, %f\n%f, %f, %f\n%f, %f, %f\n");
    printf(name,
        matrix(0, 0), matrix(0, 1), matrix(0, 2),
        matrix(1, 0), matrix(1, 1), matrix(1, 2),
        matrix(2, 0), matrix(2, 1), matrix(2, 2));
}

__device__ void printMatrix2(Eigen::Matrix2f matrix, char* name) {
    strConcat(name, ":\n%f, %f\n%f, %f\n");
    printf(name,
        matrix(0, 0), matrix(0, 1),
        matrix(1, 0), matrix(1, 1));
}

__device__ void GetTiles(
    Eigen::Vector2i imgPoint, 
    float radius, 
    Eigen::Vector2i &rectMin, 
    Eigen::Vector2i &rectMax, 
    dim3 grid) {
    rectMin[0] = min(grid.x, max(0, static_cast<int>((imgPoint[0] - radius) / BLOCK_X)));
    rectMin[1] = min(grid.y, max(0, static_cast<int>((imgPoint[1] - radius) / BLOCK_Y)));

    rectMax[0] = min(grid.x, max(0, static_cast<int>((imgPoint[0] + radius + BLOCK_X - 1) / BLOCK_X)));
    rectMax[1] = min(grid.y, max(0, static_cast<int>((imgPoint[1] + radius + BLOCK_Y - 1) / BLOCK_Y)));
}

__device__ void GetShs(Eigen::Vector3f* sh, const float* features, int idx, int M) {
    for (int j = 0; j < M; j++) {
        sh[j][0] = features[idx * M * 3 + j * 3];
        sh[j][1] = features[idx * M * 3 + j * 3 + 1];
        sh[j][2] = features[idx * M * 3 + j * 3 + 2];
    }
}

__device__ Eigen::Vector3f ComputeColor(
    int idx, int deg, 
    Eigen::Vector4f oriPoint,
    Eigen::Vector3f center, 
    const float* features, int M) {
    Eigen::Vector3f* sh = new Eigen::Vector3f[M];
    GetShs(sh, features, idx, M);

    Eigen::Vector3f dir = oriPoint.head<3>() - center;
    dir.normalize();

    Eigen::Vector3f rgb = SH_C0 * sh[0];

    if (deg > 0) {
        float x = dir[0];
        float y = dir[1];
        float z = dir[2];

        rgb = rgb - SH_C1 * y * sh[1] + SH_C1 * z * sh[2] - SH_C1 * x * sh[3];

        if (deg > 1) {
            float xx = x * x, yy = y * y, zz = z * z;
            float xy = x * y, yz = y * z, xz = x * z;
            rgb = rgb + 
                SH_C2[0] * xy * sh[4] +
                SH_C2[1] * yz * sh[5] +
                SH_C2[2] * (2.f * zz - xx - yy) * sh[6] +
                SH_C2[3] * xz * sh[7] +
                SH_C2[4] * (xx - yy) * sh[8];

            if (deg > 2) {
                rgb = rgb +
                    SH_C3[0] * y * (3.f * xx - yy) * sh[9] +
                    SH_C3[1] * xy * z * sh[10] +
                    SH_C3[2] * y * (4.f * zz - xx - yy) * sh[11] +
                    SH_C3[3] * z * (2.f * zz - 3.f * xx - 3.f * yy) * sh[12] +
                    SH_C3[4] * x * (4.f * zz - xx - yy) * sh[13] +
                    SH_C3[5] * z * (xx - yy) * sh[14] +
                    SH_C3[6] * x * (xx - 3.f * yy) * sh[15];
            }
        }
    }
    rgb = rgb.array() + 2.f;

    delete[] sh;
    //TODO: backward clamp
    return rgb.array().max(0.f);
}

__device__ Eigen::Vector2i Ndc2Pix(Eigen::Vector3f projPoint, int width, int height) {
    Eigen::Vector2i pixel;
    pixel[0] = static_cast<int>((projPoint[0] + 1) * 0.5 * width);
    pixel[1] = static_cast<int>((1 - projPoint[1]) * 0.5 * height);

    return pixel;
}

__device__ Eigen::Matrix2f ComputeCov2D(
    Eigen::Vector4f viewPoint, 
    Eigen::Matrix4f viewMatrix,
    Eigen::Matrix3f cov3D,
    float focalX, float focalY,
    float tanFovX, float tanFovY) {

    float limx = 1.3f * tanFovX;
    float limy = 1.3f * tanFovY;
    float xz = viewPoint[0] / viewPoint[2];
    float yz = viewPoint[1] / viewPoint[2];
    float x = fminf(limx, fmaxf(-limx, xz)) * viewPoint[2];
    float y = fminf(limy, fmaxf(-limy, yz)) * viewPoint[2];
    float z = viewPoint[2];

    Eigen::Matrix<float, 2, 3> J;
    J << focalX / z, 0.f, -(focalX * x) / (z * z),
        0.f, focalY / z, -(focalY * y) / (z * z);
    
    Eigen::Matrix3f W = viewMatrix.block<3, 3>(0, 0);

    Eigen::Matrix2f cov2D = J * W * cov3D * W.transpose() * J.transpose();

    cov2D(0, 0) += 0.3f;
    cov2D(1, 1) += 0.3f;

    return cov2D;
}

__device__ Eigen::Matrix3f ComputeCov3D(Eigen::Vector4f quat, Eigen::Vector3f scal) {
    Eigen::Vector4f quatNorm = quat.normalized();
    float w = quatNorm[0];
    float x = quatNorm[1];
    float y = quatNorm[2];
    float z = quatNorm[3];

    Eigen::Matrix3f R;
    R << 1.f - 2.f * y * y - 2.f * z * z, 2.f * x * y - 2.f * w * z, 2.f * x * z + 2.f * w * y,
        2.f * x * y + 2.f * w * z, 1.f - 2.f * x * x - 2.f * z * z, 2.f * y * z - 2.f * w * x,
        2.f * x * z - 2.f * w * y, 2.f * y * z + 2.f * w * x, 1.f - 2.f * x * x - 2.f * y * y;
    
    Eigen::Matrix3f cov3D;
    cov3D << scal[0], 0, 0, 0, scal[1], 0, 0, 0, scal[2];
    cov3D = R * cov3D;
    cov3D = cov3D * cov3D.transpose();

    return cov3D;
}

__global__ void DuplicateWithKeys(
    int N, 
    Eigen::Vector2i *imgXYZ, 
    float *depth,
    float *radii, 
    uint32_t* tileTouched,
    dim3 grid,
    TileDepth* tileDepthList) {

    int pid = blockDim.x * blockIdx.x + threadIdx.x;
    if (pid >= N) {
        return;
    }
    
    uint32_t off = (pid == 0) ? 0 : tileTouched[pid - 1];
    Eigen::Vector2i rectMin, rectMax;
    GetTiles(imgXYZ[pid], radii[pid], rectMin, rectMax, grid);

    for (int y = rectMin[1]; y < rectMax[1]; y++) {
        for (int x = rectMin[0]; x < rectMax[0]; x++) {
            TileDepth tileDepth;
            tileDepth.tileId = y * grid.x + x;
            tileDepth.depth = depth[pid];
            tileDepth.gaussianId = pid;

            tileDepthList[off] = tileDepth;
            off++;
        }
    }
}

__global__ void PreRenderKernel(
    const float* xyz,
    const float* rotation,
    const float* scaling,
    const float* features,

    const float near,
    const int N,
    const int width,
    const int height,
    const float* colorsPrecomp,
    const float* viewMatrix,
    const float* viewProjMatrix,
    const float* cameraCenter,
    float focalX, float focalY,
    float tanFovX, float tanFovY,
    uint32_t* tileTouched,
    const dim3 grid,
    Eigen::Vector2i* imgXYZ,
    float* depth,
    float* radii,
    float* rgb) {
    // 1.���� pid
    int pid = blockDim.x * blockIdx.x + threadIdx.x;
    if (pid >= N) {
        return;
    }

    tileTouched[pid] = 0;

    // 2.��ƽ��ü�
    Eigen::Vector4f oriPoint(xyz[pid * 3], xyz[pid * 3 + 1], xyz[pid * 3 + 2], 1.f);
    Eigen::Map<const Eigen::Matrix<float, 4, 4, Eigen::RowMajor>> viewMatrixE(viewMatrix);
    Eigen::Vector4f viewPoint = viewMatrixE * oriPoint;
    if (viewPoint[2] <= near) {
        return;
    }

    // 3.����ͶӰ
    Eigen::Map<const Eigen::Matrix<float, 4, 4, Eigen::RowMajor>> viewProjMatrixE(viewProjMatrix);
    Eigen::Vector4f projPointH = viewProjMatrixE * oriPoint;
    float w = 1.f / (projPointH[3] + 0.0000001f);
    Eigen::Vector3f projPoint(projPointH[0] * w, projPointH[1] * w, projPointH[2] * w);


    // 4.��Э�������
    Eigen::Vector4f quat(rotation[pid * 4], rotation[pid * 4 + 1], rotation[pid * 4 + 2], rotation[pid * 4 + 3]);
    Eigen::Vector3f scal(scaling[pid * 3], scaling[pid * 3 + 1], scaling[pid * 3 + 2]);
    Eigen::Matrix3f cov3D = ComputeCov3D(quat, scal);

    Eigen::Matrix2f cov2D = ComputeCov2D(viewPoint, viewMatrixE, cov3D, focalX, focalY, tanFovX, tanFovY);

    float a = cov2D(0, 0);
    float b = cov2D(0, 1);
    float c = cov2D(1, 0);
    float d = cov2D(1, 1);

    float det = (a * d - b * c);
    if (det == 0.0f) {
        return;
    }

    // 5.�����Բ
    float lambda1 = 0.5f * (a + d) + 0.5f * sqrtf((a - d) * (a - d) + 4 * b * c);
    float radius = ceilf(3.f * sqrtf(lambda1));

    // 6.tile����
    Eigen::Vector2i imgPoint = Ndc2Pix(projPoint, width, height);
    Eigen::Vector2i rectMin, rectMax;
    GetTiles(imgPoint, radius, rectMin, rectMax, grid);
    if ((rectMax[1] - rectMin[1]) * (rectMax[0] - rectMin[0]) == 0) {
        return;
    }

    if (colorsPrecomp == nullptr) {
        Eigen::Vector3f center(cameraCenter[0], cameraCenter[1], cameraCenter[2]);
        Eigen::Vector3f rgbV = ComputeColor(pid, 3, oriPoint, center, features, 16);
        rgb[pid * 3] = rgbV[0];
        rgb[pid * 3 + 1] = rgbV[1];
        rgb[pid * 3 + 2] = rgbV[2];
    }

    depth[pid] = viewPoint[2];
    radii[pid] = radius;
    imgXYZ[pid] = imgPoint;
    tileTouched[pid] = (rectMax[1] - rectMin[1]) * (rectMax[0] - rectMin[0]);
}

void printStructArray(TileDepth* array, int start, int end) {
    for (int i = start; i < end; ++i) {
        std::cout << "gaussianId: " << array[i].gaussianId << ", tileId: " << array[i].tileId << ", depth: " << array[i].depth << std::endl;
    }
}

void PreRender(
    const float* xyz, 
    const float* rotation,
    const float* scaling,
    const float* features,

    const float near, 
    const int N, 
    const int width,
    const int height,
    const float* viewMatrix,
    const float* viewProjMatrix,
    const float* cameraCenter,
    float focalX, float focalY,
    float tanFovX, float tanFovY) {

    float *d_xyz, *d_rotation, *d_scaling, *d_featrues, *d_viewMatrix, *d_viewProjMatrix, *d_cameraCenter;
    hipMalloc(&d_xyz, 3 * N * sizeof(float));
    hipMemcpy(d_xyz, xyz, 3 * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_rotation, 4 * N * sizeof(float));
    hipMemcpy(d_rotation, rotation, 4 * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_scaling, 3 * N * sizeof(float));
    hipMemcpy(d_scaling, scaling, 3 * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_featrues, 48 * N * sizeof(float));
    hipMemcpy(d_featrues, features, 48 * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_viewMatrix, 16 * sizeof(float));
    hipMemcpy(d_viewMatrix, viewMatrix, 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_viewProjMatrix, 16 * sizeof(float));
    hipMemcpy(d_viewProjMatrix, viewProjMatrix, 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_cameraCenter, 3 * sizeof(float));
    hipMemcpy(d_cameraCenter, cameraCenter, 3 * sizeof(float), hipMemcpyHostToDevice);

    uint32_t* tileTouched;
    hipMalloc(&tileTouched, N * sizeof(uint32_t));

    float* r_rgb, * r_depth, * r_radii;
    Eigen::Vector2i *r_imgXYZ;
    hipMalloc(&r_rgb, 3 * N * sizeof(float));
    hipMalloc(&r_depth, N * sizeof(float));
    hipMalloc(&r_radii, N * sizeof(float));
    hipMalloc(&r_imgXYZ, N * sizeof(Eigen::Vector2i));

    dim3 grid((width + BLOCK_X - 1) / BLOCK_X, (height + BLOCK_Y - 1) / BLOCK_Y, 1);
    dim3 block(BLOCK_X, BLOCK_Y, 1);

    PreRenderKernel <<<(N + 255) / 256, 256>>> (
        d_xyz, 
        d_rotation, 
        d_scaling, 
        d_featrues,

        near, N, 
        width, height,
        nullptr,
        d_viewMatrix, 
        d_viewProjMatrix,
        d_cameraCenter,
        focalX, focalY,
        tanFovX, tanFovY,
        tileTouched,
        grid,
        r_imgXYZ,
        r_depth,
        r_radii,
        r_rgb);

    // hipDeviceSynchronize();
    // 1��¼ÿ��tile��������GS 2��ǰ׺�ͣ���֪������װ������ 3��װ�����б� 4����

    // ���
    thrust::device_ptr<uint32_t> tileTouchedPtr(tileTouched);
    thrust::inclusive_scan(tileTouchedPtr, tileTouchedPtr + N, tileTouchedPtr);

    uint32_t numRendered;
    hipMemcpy(&numRendered, tileTouched + N - 1, sizeof(uint32_t), hipMemcpyDeviceToHost);

    TileDepth *tileDepthList;
    hipMalloc(&tileDepthList, numRendered * sizeof(TileDepth));

    // ��װ depth tileId
    DuplicateWithKeys<<<(N + 255) / 256, 256>>>(N, r_imgXYZ, r_depth, r_radii, tileTouched, grid, tileDepthList);
    
    // ����
    thrust::device_ptr<TileDepth> tileDepthListPtr(tileDepthList);
    thrust::sort(tileDepthListPtr, tileDepthListPtr + numRendered);

    /*TileDepth* test = new TileDepth[numRendered];
    hipMemcpy(test, tileDepthList, numRendered * sizeof(TileDepth), hipMemcpyDeviceToHost);
    printStructArray(test, N - 100, N);*/
 

    //delete[] test;
    hipFree(d_xyz);
    hipFree(d_rotation);
    hipFree(d_scaling);
    hipFree(d_featrues);
    hipFree(d_viewMatrix);
    hipFree(d_viewProjMatrix);
    hipFree(d_cameraCenter);
    hipFree(tileTouched);
    hipFree(r_imgXYZ);
    hipFree(r_depth);
    hipFree(r_radii);
    hipFree(r_rgb);
    hipFree(tileDepthList);
}