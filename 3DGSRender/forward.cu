#include "hip/hip_runtime.h"
#include "forward.h"

__global__ void PreRenderKernel(
    const float* xyz,
    //const float* rotation,
    //const float* scaling,
    //const float* opacity,
    //const float* features,

    const float near,
    const int N,
    //const int width,
    //const int height,
    const float* viewMatrix,
    const float* viewProjMatrix
    
    ) {
    // 1.���� pid    
    int pid = blockDim.x * blockIdx.x + threadIdx.x;
    if (pid >= N) {
        return;
    }
    // 2.��ƽ��ü�
    Eigen::Vector4f oriPoint(xyz[pid * 3], xyz[pid * 3 + 1], xyz[pid * 3 + 2], 1.f);
    Eigen::Map<const Eigen::Matrix<float, 4, 4, Eigen::RowMajor>> viewMatrixE(viewMatrix);
    Eigen::Vector4f viewPoint = viewMatrixE * oriPoint;
    if (viewPoint[2] <= near) {
        return;
    }

    // 3.����ͶӰ
    Eigen::Map<const Eigen::Matrix<float, 4, 4, Eigen::RowMajor>> viewProjMatrixE(viewProjMatrix);
    Eigen::Vector4f projPoint = viewProjMatrixE * oriPoint;


   /* if (pid == 1) {
        printf("oriPoint: %f,%f,%f,%f\n", oriPoint[0], oriPoint[1], oriPoint[2], oriPoint[3]);
        printf("viewPoint: %f,%f,%f,%f\n", viewPoint[0], viewPoint[1], viewPoint[2], viewPoint[3]);
    }*/
    
    // 4.�������
}

void  PreRender(
    const float* xyz, 
    
    const float near, 
    const int N, 
    const float* viewMatrix,
    const float* viewProjMatrix
) {
    float *d_xyz, *d_viewMatrix, *d_viewProjMatrix;
    hipMalloc(&d_xyz, 3 * N * sizeof(float));
    hipMemcpy(d_xyz, xyz, 3 * N * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_viewMatrix, 16 * sizeof(float));
    hipMemcpy(d_viewMatrix, viewMatrix, 16 * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_viewProjMatrix, 16 * sizeof(float));
    hipMemcpy(d_viewProjMatrix, viewProjMatrix, 16 * sizeof(float), hipMemcpyHostToDevice);

    PreRenderKernel <<<(N + 255) / 256, 256 >>> (d_xyz, near, N, d_viewMatrix, d_viewProjMatrix);

    hipFree(d_xyz);
    hipFree(d_viewMatrix);
    hipFree(d_viewProjMatrix);
}